#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

#define O_SPARTA 0x01

#define CHARMAP "abcdefghijklmnopqrstuvwxyz"\
		"ABCDEFGHIJKLMNOPQRSTUVWXYZ"\
		" "

#define RANDBETWEEN(A,B) A + rand()/(RAND_MAX/(B - A))
#define CHANCE(A) rand() < A * RAND_MAX

static char* target = "Hello world";
static size_t el_sz;
static size_t total_sz;
static char options = 0;
static unsigned int pop_size = 2048;
static unsigned int challengers = 3;
static float elitism = .1;
static float mutation = .25;

static char
rndchr(char* map)
{
	return *(map+RANDBETWEEN(0, strlen(map)));
}

static char
randchar()
{
	return rndchr(CHARMAP);
}

static char*
rndstr(char* map, size_t strsize)
{
	char *result = malloc(strsize);
	size_t i;

	for (i = 0; i < strsize; i++) {
		*(result+i) = rndchr(map);
	};

	return result;
}

static int
fitness(char* str, char* gauge, size_t n)
{
	int i;
	int result = 0;

	for (i = n-1; i >= 0; i--) {
		result += abs(str[i]-gauge[i]);
	}

	return result;
}

static int
_fitness(char* str)
{
	return fitness(str, target, el_sz);
}

static int
fit_cmp(const void *el1, const void *el2)
{
	int a = _fitness((char*)el1);
	int b = _fitness((char*)el2);

	if (a > b) return 1;
	if (a < b) return -1;
	return 0;
}

static void
mutate(char *p)
{
	*(p + RANDBETWEEN(0, el_sz)) = randchar();
}

static char*
rnd_el(char *p)
{
	unsigned int top = pop_size;

	if ((options & O_SPARTA) == O_SPARTA) {
		top = pop_size * elitism;
	}

	return p + el_sz * (int)(RANDBETWEEN(0, top));
}

static char*
trnmnt(char *p)
{
	size_t i;
	char* winner = rnd_el(p);
	char* challenger;
	int f1 = _fitness(winner);
	int f2;

	for (i = challengers; i > 0; i--) {
		challenger = rnd_el(p);
		f2 = _fitness(challenger);
		if (f2 < f1) {
			f1 = f2;
			winner = challenger;
		}
	}

	return winner;
}

static void
mate(char *p, char *buffer)
{
	char *a, *b;
	size_t i, pivot;
	size_t skip = (size_t)(elitism * pop_size) * el_sz;
	memcpy(buffer, p, total_sz);

	for (i = skip; i <= total_sz-el_sz; i += el_sz) {
		a = trnmnt(p);
		b = trnmnt(p);
		pivot = RANDBETWEEN(0, el_sz);

		strncpy(buffer + i, a, el_sz);
		strncpy(buffer + i, b, pivot);

		if (CHANCE(mutation)) { mutate(buffer + i); }

		if (i < total_sz - el_sz) {
			i += el_sz;
			strncpy(buffer + i, b, el_sz);
			strncpy(buffer + i, a, pivot);
			if (CHANCE(mutation)) { mutate(buffer + i); }
		}
	}

	memcpy(p, buffer, total_sz);
}

static void
run_tests(void)
{
	assert(000 == fitness("Hello world",  "Hello world", 11));
	printf("Tests passed.\n\n");
}

static void
print_usage(char *self)
{
	printf("Usage: %s [-t] [-s] [-h] [-p SIZE] [-c COUNT] [-e RATIO] [-m RATIO] [-i STRING]\n", self);
	printf("	-t:		run tests\n");
	printf("	-s:		Sparta! mode (Only elite can mate)\n");
	printf("	-h:		Show this help\n");
	printf("	-p SIZE:	Population size\n");
	printf("	-c COUNT:	Challengers count for mate tournament\n");
	printf("	-e RATIO:	Elitism ratio\n");
	printf("	-m RATIO:	Mutation ratio\n");
	printf("	-i STRING:	search this instead of \"Hello, World!\"\n");
}

static void
check_params()
{
	if ((options & O_SPARTA) == O_SPARTA
		&& ((int)(pop_size * elitism) == 0)) {
		printf("You have not enough spartans.\n");
		exit(1);
	}
}

int main(int argc, char **argv)
{
	int i = 0;
	int bestfit = RAND_MAX;
	int opt;
	srand((unsigned int)time(NULL));

	while((opt = getopt(argc, argv, "tshi:p:e:m:c:")) != -1) {
		switch (opt) {
		case 't':
			run_tests();
			break;
		case 's':
			options |= O_SPARTA;
			break;
		case 'i':
			target = optarg;
			break;
		case 'p':
			pop_size = atoi(optarg);
			break;
		case 'c':
			challengers = atoi(optarg);
			break;
		case 'e':
			elitism = atof(optarg);
			break;
		case 'm':
			mutation = atof(optarg);
			break;
		case 'h':
		default:
			print_usage(argv[0]);
			exit(1);
		}
	}

	check_params();

	el_sz = strlen(target);
	total_sz = pop_size * el_sz;
	char *p = rndstr(CHARMAP, total_sz);
	char *b = malloc(total_sz);

	while (bestfit) {
		qsort(p, pop_size, el_sz, fit_cmp);
		i += 1;

		if (bestfit != _fitness(p)) {
			bestfit = _fitness(p);
			printf("[%03d] Best: (%04d)\t%.*s\n", i,
				bestfit, (int)el_sz, p);
		}

		mate(p, b);
	}

	free(p);
	free(b);

	return 0;
}