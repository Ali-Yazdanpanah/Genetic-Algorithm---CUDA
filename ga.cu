

#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// Macro for checking cuda errors following a cuda launch or api call
#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }

#define O_SPARTA 0x01

#define CHARMAP                                                                \
  "abcdefghijklmnopqrstuvwxyz"                                                 \
  "ABCDEFGHIJKLMNOPQRSTUVWXYZ"                                                 \
  " "

#define RANDBETWEEN(A, B) A + rand() / (RAND_MAX / (B - A))
#define CHANCE(A) rand() < A *RAND_MAX

static char *target = "Hello world";
static size_t el_sz;
static size_t total_sz;
static char options = 0;
static unsigned int pop_size = 2048;
static unsigned int challengers = 3;
static float elitism = .1;
static float mutation = .25;

static char rndchr(char *map) { return *(map + RANDBETWEEN(0, strlen(map))); }

static char randchar() { return rndchr(CHARMAP); }

static char *rndstr(char *map, size_t strsize) {
  char *result = (char *)malloc(strsize * sizeof(char));
  size_t i;

  for (i = 0; i < strsize; i++) {
    *(result + i) = rndchr(map);
  };

  return result;
}

static int fitness(char *str, char *gauge, size_t n) {
  int i;
  int result = 0;

  for (i = n - 1; i >= 0; i--) {
    result += abs(str[i] - gauge[i]);
  }

  return result;
}

static int _fitness(char *str) { return fitness(str, target, el_sz); }

static int fit_cmp(const void *el1, const void *el2) {
  int a = _fitness((char *)el1);
  int b = _fitness((char *)el2);

  if (a > b)
    return 1;
  if (a < b)
    return -1;
  return 0;
}

static void mutate(char *p) { *(p + RANDBETWEEN(0, el_sz)) = randchar(); }

static char *rnd_el(char *p) {
  unsigned int top = pop_size;

  if ((options & O_SPARTA) == O_SPARTA) {
    top = pop_size * elitism;
  }

  return p + el_sz * (int)(RANDBETWEEN(0, top));
}

static char *trnmnt(char *p) {
  size_t i;
  char *winner = rnd_el(p);
  char *challenger;
  int f1 = _fitness(winner);
  int f2;

  for (i = challengers; i > 0; i--) {
    challenger = rnd_el(p);
    f2 = _fitness(challenger);
    if (f2 < f1) {
      f1 = f2;
      winner = challenger;
    }
  }

  return winner;
}

static void mate(char *p, char *buffer) {
  char *a, *b;
  size_t i, pivot;
  size_t skip = (size_t)(elitism * pop_size) * el_sz;
  memcpy(buffer, p, total_sz);

  for (i = skip; i <= total_sz - el_sz; i += el_sz) {
    a = trnmnt(p);
    b = trnmnt(p);
    pivot = RANDBETWEEN(0, el_sz);

    strncpy(buffer + i, a, el_sz);
    strncpy(buffer + i, b, pivot);

    if (CHANCE(mutation)) {
      mutate(buffer + i);
    }

    if (i < total_sz - el_sz) {
      i += el_sz;
      strncpy(buffer + i, b, el_sz);
      strncpy(buffer + i, a, pivot);
      if (CHANCE(mutation)) {
        mutate(buffer + i);
      }
    }
  }

  memcpy(p, buffer, total_sz);
}

static void run_tests(void) {
  assert(000 == fitness("Hello world", "Hello world", 11));
  printf("Tests passed.\n\n");
}

static void print_usage(char *self) {
  printf("Usage: %s [-t] [-s] [-h] [-p SIZE] [-c COUNT] [-e RATIO] [-m RATIO] "
         "[-i STRING]\n",
         self);
  printf("	-t:		run tests\n");
  printf("	-s:		Sparta! mode (Only elite can mate)\n");
  printf("	-h:		Show this help\n");
  printf("	-p SIZE:	Population size\n");
  printf("	-c COUNT:	Challengers count for mate tournament\n");
  printf("	-e RATIO:	Elitism ratio\n");
  printf("	-m RATIO:	Mutation ratio\n");
  printf("	-i STRING:	search this instead of \"Hello, World!\"\n");
}

static void check_params() {
  if ((options & O_SPARTA) == O_SPARTA && ((int)(pop_size * elitism) == 0)) {
    printf("You have not enough spartans.\n");
    exit(1);
  }
}

void swap(int a, int b, int *fitness, char *p, int el_sz) {
  int temp = fitness[a];
  fitness[a] = fitness[b];
  fitness[b] = temp;
  char tempchar;
  for (int i = 0; i < el_sz; i++) {
    tempchar = p[a + i];
    p[a + i] = p[b + i];
    p[b + i] = tempchar;
  }
}

int partition(int *fitness, int low, int high, char *p, int el_sz) {
  int pivot = fitness[high]; // pivot
  int i = (low - 1);         // Index of smaller element
  for (int j = low; j <= high - 1; j++) {
    // If current element is smaller than the pivot
    if (fitness[j] < pivot) {
      i++; // increment index of smaller element
      swap(i, j, fitness, p, el_sz);
    }
  }
  swap(i + 1, high, fitness, p, el_sz);
  return (i + 1);
}

void quickSort(int *fitness, int low, int high, char *p, int el_sz) {
  if (low < high) {
    /* pi is partitioning index, arr[p] is now
at right place */
    int pi = partition(fitness, low, high, p, el_sz);
    // Separately sort elements before
    // partition and after partition
    quickSort(fitness, low, pi - 1, p, el_sz);
    quickSort(fitness, pi + 1, high, p, el_sz);
  }
}

void calculate_fitness(int *fitness, char *p, int el_sz, int total_sz,
                       char *gauge) {
  int result;
  int j = 0;
  for (int i = 0; i < total_sz; i += el_sz) {
    result = 0;
    for (int k = 0; k < el_sz; k++) {
      result += abs(p[i + k] - gauge[k]);
    }
    fitness[j] = result;
    j++;
  }
}

__global__ void fitness_kernel(int *fitness, char *p, int el_sz, int total_sz,
                               char *gauge) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < total_sz) {
    int result = 0;
    for (int k = 0; k < el_sz; k++) {
      result += abs(p[idx + k] - gauge[k]);
    }
    fitness[idx] = result;
  }
}

int main(int argc, char **argv) {
  struct timeval start, end;
  int i = 0;
  int bestfit = RAND_MAX;
  int opt;
  srand((unsigned int)time(NULL));
  while ((opt = getopt(argc, argv, "tshi:p:e:m:c:")) != -1) {
    switch (opt) {
    case 't':
      run_tests();
      break;
    case 's':
      options |= O_SPARTA;
      break;
    case 'i':
      target = optarg;
      break;
    case 'p':
      pop_size = atoi(optarg);
      break;
    case 'c':
      challengers = atoi(optarg);
      break;
    case 'e':
      elitism = atof(optarg);
      break;
    case 'm':
      mutation = atof(optarg);
      break;
    case 'h':
    default:
      print_usage(argv[0]);
      exit(1);
    }
  }

  check_params();

  el_sz = strlen(target);
  total_sz = pop_size * el_sz;
  char *p = (char *)rndstr(CHARMAP, total_sz);
  char *b = (char *)malloc(total_sz * sizeof(char));
  char *d_p;
  int *d_fitness;
  char *d_target;
  int *fitness = (int *)malloc(pop_size * sizeof(int));
  int grids;
  if (pop_size > 64) {
    grids = pop_size / 64 + 1;
  }
  dim3 grid_dime(grids, 1, 1);
  dim3 block_dime(64, 1, 1);
  CUDA_CHECK_RETURN(hipMalloc((void **)&d_p, sizeof(char) * total_sz));
  CUDA_CHECK_RETURN(hipMalloc((void **)&d_fitness, sizeof(int) * pop_size));
  CUDA_CHECK_RETURN(hipMalloc((void **)&d_target, sizeof(char) * el_sz));

  CUDA_CHECK_RETURN(hipMemcpy(d_target, target, sizeof(char) * el_sz,
                               hipMemcpyHostToDevice));
  gettimeofday(&start, NULL);
  while (bestfit) {
    CUDA_CHECK_RETURN(
        hipMemcpy(d_p, p, sizeof(char) * total_sz, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_fitness, fitness, sizeof(int) * pop_size,
                                 hipMemcpyHostToDevice));
    printf("%d \n", i);
    fitness_kernel<<<grid_dime, block_dime>>>(d_fitness, d_p, el_sz, total_sz,
                                              d_target);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipMemcpy(fitness, d_fitness, sizeof(int) * pop_size,
                                 hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(
        hipMemcpy(p, d_p, sizeof(char) * total_sz, hipMemcpyDeviceToHost));
    quickSort(fitness, 0, pop_size - 1, p, el_sz);
    i += 1;
    if (bestfit != fitness[0]) {
      bestfit = fitness[0];
      if (bestfit == 0)
        printf("[%03d] result found \n", i);
    }
    mate(p, b);
  }
  gettimeofday(&end, NULL);
  double diffDouble =
      (end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec);
  printf("Execution took : %.4fms\n", diffDouble);
  free(p);
  free(b);

  return 0;
}
